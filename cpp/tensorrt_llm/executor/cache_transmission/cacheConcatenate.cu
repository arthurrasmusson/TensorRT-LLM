#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-2024 NVIDIA CORPORATION &
 * AFFILIATES. All rights reserved. SPDX-License-Identifier: NVIDIA TensorRT
 * Source Code License Agreement
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include "../dataTransceiverState.h"
#include "cacheConcatenate.h"
#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaFp8Utils.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/dataType.h"
#include "tensorrt_llm/common/mpiUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/executor/tensor.h"
#include "tensorrt_llm/executor/types.h"
#include "tensorrt_llm/runtime/bufferManager.h"
#include "tensorrt_llm/runtime/iBuffer.h"
#include "tensorrt_llm/runtime/iTensor.h"
#include <NvInferRuntimeBase.h>
#include <cstdint>
#include <sstream>
#include <string>
#include <vector>

namespace tensorrt_llm::executor::kv_cache
{

// inputBlockNums:[ outputBlockNum , inputRanks.size]
// [PP,TP]
TargetRanksInfo targetIRanks(
    kv_cache::CacheState const& iCacheState, kv_cache::CacheState const& oCacheState, int oRank)
{
    int iPPNum = iCacheState.getParallelConfig().mPipelineParallelism; // TODO:
    int oPPNum = oCacheState.getParallelConfig().mPipelineParallelism;
    int oNbKvHeads = oCacheState.getModelConfig().mNbKvHeadsPerLayer[0];
    int oNbLayers = oCacheState.getModelConfig().mNbKvHeadsPerLayer.size() / oPPNum;
    int iNbKvHeads = iCacheState.getModelConfig().mNbKvHeadsPerLayer[0];
    int iNbLayers = iCacheState.getModelConfig().mNbKvHeadsPerLayer.size() / iPPNum;
    int oTpRank = oRank % oCacheState.getParallelConfig().mTensorParallelism;
    int oPpRank = oRank / oCacheState.getParallelConfig().mTensorParallelism;
    int startHeadId = oTpRank * oNbKvHeads;
    int endHeadId = (oTpRank + 1) * oNbKvHeads;
    int startLayerId = oPpRank * oNbLayers;
    int endLayerId = (oPpRank + 1) * oNbLayers;
    int iTpRankStart = startHeadId / iNbKvHeads;
    int iTpRankEndInclude = (endHeadId - 1) / iNbKvHeads;
    int iPpRankStart = startLayerId / iNbLayers;
    int iPpRankEndInclude = (endLayerId - 1) / iNbLayers;

    int iTPNum = iCacheState.getParallelConfig().mTensorParallelism;
    std::vector<int> retRanks;

    for (int i = iTpRankStart; i <= iTpRankEndInclude; i++)
    {
        for (int j = iPpRankStart; j <= iPpRankEndInclude; j++)
        {
            int irank = j * iTPNum + i;
            retRanks.push_back(irank);
        }
    }
    // [tp ,pp]  order
    int mDomainPPSize = iPpRankEndInclude - iPpRankStart + 1;
    int mDomainTPSize = iTpRankEndInclude - iTpRankStart + 1;
    return {mDomainPPSize, mDomainTPSize, std::move(retRanks)};
}

template <typename T>
struct BlockInfo
{

    T* data;

    int startTokenId;
    int tokensPerBlock;

    int startHeadId;
    int headsPerBlock;

    int startLayerId;
    int layersPerBlock;

    int dimsPerHead;
    size_t offset; // (data-offset)[idx]

    __forceinline__ __device__ __host__ T* getKblockPtr(int layerid)
    {
        // return layerid- startLayerId
        return data + (layerid * 2) * headsPerBlock * tokensPerBlock * dimsPerHead;
    }

    __forceinline__ __device__ __host__ T* getVblockPtr(int layerid)
    {
        return data + (layerid * 2 + 1) * headsPerBlock * tokensPerBlock * dimsPerHead;
    }

    __forceinline__ __device__ __host__ T* getKDimsPtr(int layerid, int headid, int tokenid)
    {
        return data + (layerid * 2) * headsPerBlock * tokensPerBlock * dimsPerHead
            + headid * tokensPerBlock * dimsPerHead + tokenid * dimsPerHead;
    }

    __forceinline__ __device__ __host__ T const* getKDimsPtr(int layerid, int headid, int tokenid) const
    {
        return data + (layerid * 2) * headsPerBlock * tokensPerBlock * dimsPerHead
            + headid * tokensPerBlock * dimsPerHead + tokenid * dimsPerHead;
    }

    __forceinline__ __device__ __host__ T* getVDimsPtr(int layerid, int headid, int tokenid)
    {
        return data + (layerid * 2 + 1) * headsPerBlock * tokensPerBlock * dimsPerHead
            + headid * tokensPerBlock * dimsPerHead + tokenid * dimsPerHead;
    }

    __forceinline__ __device__ __host__ T const* getVDimsPtr(int layerid, int headid, int tokenid) const
    {
        return data + (layerid * 2 + 1) * headsPerBlock * tokensPerBlock * dimsPerHead
            + headid * tokensPerBlock * dimsPerHead + tokenid * dimsPerHead;
    }

    std::string to_string()
    {
        std::stringstream ss;
        ss << "{data ptr: " << data << "startTokenId: " << startTokenId << "tokensPerBlock:  " << tokensPerBlock
           << " startHeadId: " << startHeadId << "headsPerBlock: " << headsPerBlock << "startLayerId:" << startLayerId
           << "layersPerBlock: " << layersPerBlock << "dimsPerHead: " << dimsPerHead << " offset: " << offset << "}";
        return ss.str();
    }
};

// refer blockPtr

// Block shape [ head,tokens,dimsPerHead]
//  CacheBlock [numLayers,2,mBlockSize] . BlockSize[

// kV  and copy

// note k and v not continuous

__forceinline__ __device__ int getInputBlockId(int outputBlockId, int headId, int layerId, int inputBlockNumEachOutput,
    int headNumPerBlock, int layerNumPerBlock, int headNumInputModel, int layerNumInputModel)
{

    int offset = outputBlockId * inputBlockNumEachOutput;

    int layerOffset = layerId / layerNumPerBlock;

    int headOffset = headId / headNumPerBlock;

    int headBlockNum = headNumInputModel / headNumPerBlock;
    return offset + layerOffset * headBlockNum + headOffset;
}

// subWarpSize*subWarpGroupSize
template <typename T, int subWarpSize, int subWarpNumInGroup, int vecSizeByte>
__global__ void splitAndConcatenateBlocksKernel(BlockInfo<T> const* iBlockInfo, BlockInfo<T>* oBlockInfo, int iBlockNum,
    int iNumBlockEachO, int oBlockNum, int headNumInputModel, int layerNumInputModel, int iHeadsPerBlock,
    int iLayersPerBlock)
{

    // for blockDim.y for output_blockNum
    // blockDim.x for layer

    // wraps for heads*tokens
    // threads for dimsPerHead

    // input_id can be decided by outputid,layerid,headid
    // cuda blockNum layers*oBlockNum

    int const subWarpId = threadIdx.x / subWarpSize;
    int const laneId = threadIdx.x % subWarpSize;
    int const subWarpNum = blockDim.x / subWarpSize;
    int const subWarpGroupId = subWarpId / subWarpNumInGroup; //
    int const subWarpGroupNum = subWarpNum / subWarpNumInGroup;
    int const subWarpIdInGroup = subWarpId % subWarpNumInGroup;
    static_assert(vecSizeByte >= sizeof(T));
    int constexpr numElePerThread = vecSizeByte / sizeof(T);
    // using VecType = typename common::packed_as<T,numElePerThread>::type;
    using VecType = typename common::BytesToType<vecSizeByte>::type;
#pragma unroll 1
    for (int oBlockId = blockIdx.y; oBlockId < oBlockNum; oBlockId += gridDim.y)
    {
        int oLayerNum = oBlockInfo[oBlockId].layersPerBlock;
        int headNum = oBlockInfo[oBlockId].headsPerBlock;
        int tokenNum = oBlockInfo[oBlockId].tokensPerBlock;
        int dimsPerHead = oBlockInfo[oBlockId].dimsPerHead;
#pragma unroll 1

        for (int layerid = blockIdx.x; layerid < oLayerNum; layerid += gridDim.x)
        {
#pragma unroll 1

            for (int headId = subWarpGroupId; headId < headNum; headId += subWarpGroupNum)
            {
                int const targetHeadId = oBlockInfo[oBlockId].startHeadId + headId;
                int const targetLayerId = oBlockInfo[oBlockId].startLayerId + layerid;

                int const iBlockId = getInputBlockId(oBlockId, targetHeadId, targetLayerId, iNumBlockEachO,
                    iHeadsPerBlock, iLayersPerBlock, headNumInputModel, layerNumInputModel);
                int const iLayerId = targetLayerId % iLayersPerBlock;
                int const iHeadId = targetHeadId % iHeadsPerBlock;
#pragma unroll 1

                for (int tokenId = subWarpIdInGroup; tokenId < tokenNum; tokenId += subWarpNumInGroup)
                {

                    T* oKPtr = oBlockInfo[oBlockId].getKDimsPtr(layerid, headId, tokenId);
                    T const* iKPtr = iBlockInfo[iBlockId].getKDimsPtr(iLayerId, iHeadId, tokenId);
                    T* oVPtr = oBlockInfo[oBlockId].getVDimsPtr(layerid, headId, tokenId);
                    T const* iVPtr = iBlockInfo[iBlockId].getVDimsPtr(iLayerId, iHeadId, tokenId);
#pragma unroll 1

                    for (int channelId = laneId * numElePerThread; channelId < dimsPerHead;
                         channelId += (subWarpSize * numElePerThread))
                    {

                        common::copy<vecSizeByte>(iKPtr + channelId, oKPtr + channelId);
                        common::copy<vecSizeByte>(iVPtr + channelId, oVPtr + channelId);
                    }
                }
            }
        }
    }
}

template <typename T>
void concatenateKVCache(runtime::ITensor::SharedPtr* inputBlocks, int inputBlockNum, std::vector<int> const& inputRanks,
    kv_cache::CacheState const& iCacheState, runtime::ITensor::SharedPtr* outputBlocks, int outputBlockNum, int oRank,
    kv_cache::CacheState const& oCacheState, runtime::BufferManager const& bufferManager)

{

    TLLM_CHECK_WITH_INFO(!inputRanks.empty(), "input should not be empty!");
    TLLM_CHECK_WITH_INFO(
        inputBlockNum == outputBlockNum * inputRanks.size(), "inputBlockNum==outputBlockNum*inputRanks.size()");

    TLLM_CHECK(inputRanks == targetIRanks(iCacheState, oCacheState, oRank).mIRanks);
    int const inputAllRankNum
        = iCacheState.getParallelConfig().mPipelineParallelism * iCacheState.getParallelConfig().mTensorParallelism;
    std::vector<BlockInfo<T>> blockInfos(outputBlockNum * inputAllRankNum + outputBlockNum);

    auto fillBlockInfo = [](kv_cache::CacheState const& cacheState, runtime::ITensor::SharedPtr buffer, int rank)
    {
        int tpRank = rank % cacheState.getParallelConfig().mTensorParallelism;
        int ppRank = rank / cacheState.getParallelConfig().mTensorParallelism;
        int ppNum = cacheState.getParallelConfig().mPipelineParallelism;
        int headsPerBlock = cacheState.getModelConfig().mNbKvHeadsPerLayer[0];
        int layersPerBlock = cacheState.getModelConfig().mNbKvHeadsPerLayer.size() / ppNum; //  TODO:need  / PPSize?

        int tokensPerBlock = cacheState.getModelConfig().mTokensPerBlock;
        int dimsPerBlock = cacheState.getModelConfig().mSizePerHead;
        int startHead = tpRank * headsPerBlock;
        int startLayer = ppRank * layersPerBlock;
        // TODO:just ignore start Tokenid
        int startTokenId = 0;
        T* data = static_cast<T*>(buffer->data());
        return BlockInfo<T>{
            data, startTokenId, tokensPerBlock, startHead, headsPerBlock, startLayer, layersPerBlock, dimsPerBlock, 0};
    };
    // fill blcokInfo from CacheState and inputBlocks
    for (int oi = 0; oi < outputBlockNum; oi++)
    {
        int iRankNum = inputRanks.size();
        for (int i = 0; i < iRankNum; i++)
        {
            int iRank = inputRanks[i];
            blockInfos[oi * inputAllRankNum + iRank]
                = fillBlockInfo(iCacheState, inputBlocks[oi * iRankNum + i], iRank);
        }

        blockInfos[outputBlockNum * inputAllRankNum + oi] = fillBlockInfo(oCacheState, outputBlocks[oi], oRank);
    }
    runtime::BufferManager::IBufferPtr blockInfosDeviceBuffer
        = bufferManager.gpu(sizeof(BlockInfo<T>) * (blockInfos.size()), nvinfer1::DataType::kUINT8);
    bufferManager.copy((blockInfos.data()), *blockInfosDeviceBuffer, runtime::MemoryType::kCPU);

    BlockInfo<T>* iBlockInfoDevice = static_cast<BlockInfo<T>*>(blockInfosDeviceBuffer->data());

    BlockInfo<T>* oBlockInfoDevice = iBlockInfoDevice + outputBlockNum * inputAllRankNum;

    constexpr int subWarpSize = 8;
    constexpr int subWarpNumInGroup = 8;
    int blockDimx = 128;
    int oPpNum = oCacheState.getParallelConfig().mPipelineParallelism;
    int iPpNum = iCacheState.getParallelConfig().mPipelineParallelism;
    unsigned int gridDimx = oCacheState.getModelConfig().mNbKvHeadsPerLayer.size() / oPpNum;
    unsigned int gridDimy = outputBlockNum;

    dim3 gridDim{gridDimx, gridDimy};
    int const headsInputModel
        = iCacheState.getModelConfig().mNbKvHeadsPerLayer[0] * iCacheState.getParallelConfig().mTensorParallelism;
    int const layersInputModel = iCacheState.getModelConfig().mNbKvHeadsPerLayer.size();
    int const iHeadsPerBlock = iCacheState.getModelConfig().mNbKvHeadsPerLayer[0];
    int const iLayersPerBlock = iCacheState.getModelConfig().mNbKvHeadsPerLayer.size() / iPpNum;
    int const sizePerHead = oCacheState.getModelConfig().mSizePerHead;
    int const remainder = sizePerHead * sizeof(T) % 16;
    switch (remainder)
    {
    case 0:
    {
        splitAndConcatenateBlocksKernel<T, subWarpSize, subWarpNumInGroup, 16>
            <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(iBlockInfoDevice, oBlockInfoDevice,
                outputBlockNum * inputAllRankNum, inputAllRankNum, outputBlockNum, headsInputModel, layersInputModel,
                iHeadsPerBlock, iLayersPerBlock);
        break;
    }
    case 8:
    {
        splitAndConcatenateBlocksKernel<T, subWarpSize, subWarpNumInGroup, 8>
            <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(iBlockInfoDevice, oBlockInfoDevice,
                outputBlockNum * inputAllRankNum, inputAllRankNum, outputBlockNum, headsInputModel, layersInputModel,
                iHeadsPerBlock, iLayersPerBlock);
        break;
    }
    case 4:
    case 12:
    {
        if constexpr (sizeof(T) <= 4)
        {
            splitAndConcatenateBlocksKernel<T, subWarpSize, subWarpNumInGroup, 4>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(iBlockInfoDevice, oBlockInfoDevice,
                    outputBlockNum * inputAllRankNum, inputAllRankNum, outputBlockNum, headsInputModel,
                    layersInputModel, iHeadsPerBlock, iLayersPerBlock);
            break;
        }
    }
    case 2:
    case 6:
    case 10:
    case 14:
    {
        if constexpr (sizeof(T) <= 2)
        {

            splitAndConcatenateBlocksKernel<T, subWarpSize, subWarpNumInGroup, 2>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(iBlockInfoDevice, oBlockInfoDevice,
                    outputBlockNum * inputAllRankNum, inputAllRankNum, outputBlockNum, headsInputModel,
                    layersInputModel, iHeadsPerBlock, iLayersPerBlock);
            break;
        }
    }
    default:
    {
        if constexpr (sizeof(T) <= 1)
        {
            splitAndConcatenateBlocksKernel<T, subWarpSize, subWarpNumInGroup, 1>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(iBlockInfoDevice, oBlockInfoDevice,
                    outputBlockNum * inputAllRankNum, inputAllRankNum, outputBlockNum, headsInputModel,
                    layersInputModel, iHeadsPerBlock, iLayersPerBlock);
        }
        else
        {
            TLLM_THROW(" concatenateKVCacheDispatch no support data type error");
        }
    }
    }
}

void concatenateKVCacheDispatch(runtime::ITensor::SharedPtr* inputBlocks, int inputBlockNum,
    std::vector<int> const& inputRanks, kv_cache::CacheState const& iCacheState,
    runtime::ITensor::SharedPtr* outputBlocks, int outputBlockNum, int oRanks, kv_cache::CacheState const& oCacheState,
    runtime::BufferManager const& bufferManager)
{
    auto dataType = outputBlocks[0]->getDataType();
    int dataSize = tensorrt_llm::common::getDTypeSize(dataType);

    switch (dataSize)
    {
    case 8:
    {
        concatenateKVCache<double>(inputBlocks, inputBlockNum, inputRanks, iCacheState, outputBlocks, outputBlockNum,
            oRanks, oCacheState, bufferManager);
        break;
    }
    case 4:
    {
        concatenateKVCache<float>(inputBlocks, inputBlockNum, inputRanks, iCacheState, outputBlocks, outputBlockNum,
            oRanks, oCacheState, bufferManager);
        break;
    }
    case 2:
    {

        concatenateKVCache<half>(inputBlocks, inputBlockNum, inputRanks, iCacheState, outputBlocks, outputBlockNum,
            oRanks, oCacheState, bufferManager);
        break;
    }

    case 1:
    {

        concatenateKVCache<uint8_t>(inputBlocks, inputBlockNum, inputRanks, iCacheState, outputBlocks, outputBlockNum,
            oRanks, oCacheState, bufferManager);
        break;
    }

    default:
    {
        TLLM_THROW(" concatenateKVCacheDispatch no support");
    }
    }
}

nvinfer1::Dims makeShapeFromCacheState(kv_cache::CacheState const& cacheState)
{

    long blockSize = cacheState.getModelConfig().mNbKvHeadsPerLayer[0] * cacheState.getModelConfig().mTokensPerBlock
        * cacheState.getModelConfig().mSizePerHead;
    int PpNum = cacheState.getParallelConfig().mPipelineParallelism;
    return runtime::ITensor::makeShape(
        {static_cast<long>(cacheState.getModelConfig().mNbKvHeadsPerLayer.size() / PpNum), 2, blockSize});
}

// Block shape [ head,tokens,dimsPerHead]
//  CacheBlock [numLayers,2,mBlockSize] .

//[outputSplitCaches,numLayers,2,head,tokens_per_block,dimsPerHead]
// tokens maybe large, so We

// subWarpSize*subWarpGroupSize
template <typename T, int subWarpSize, int subWarpNumInGroup, int vecSizeByte>
__global__ void splitKVCacheKernel(T const** __restrict__ inputBlocks, T** __restrict__ outputCaches,
    int tokensPerBlock, int numLayers, int headNum, int dimsPerHead, int inputBlockNum, int DomainPPSize,
    int DomainTPSize, int layerNumDomainPP, int headNumDomainTP)
{

    int const subWarpId = threadIdx.x / subWarpSize;
    int const laneId = threadIdx.x % subWarpSize;
    int const subWarpNum = blockDim.x / subWarpSize;
    int const subWarpGroupId = subWarpId / subWarpNumInGroup; //
    int const subWarpGroupNum = subWarpNum / subWarpNumInGroup;
    int const subWarpIdInGroup = subWarpId % subWarpNumInGroup;
    static_assert(vecSizeByte >= sizeof(T));
    int constexpr numElePerThread = vecSizeByte / sizeof(T);
    using VecType = typename common::BytesToType<vecSizeByte>::type;
#pragma unroll 1

    for (int blockId = blockIdx.y; blockId < inputBlockNum; blockId += gridDim.y)
    {
#pragma unroll 1

        for (int layerId = blockIdx.x; layerId < numLayers; layerId += gridDim.x)
        {
#pragma unroll 1

            for (int headId = subWarpGroupId; headId < headNum; headId += subWarpGroupNum)
            {

                T const* inputBlockPtr = inputBlocks[blockId];
                T const* kInputPtr = inputBlockPtr + layerId * 2 * headNum * tokensPerBlock * dimsPerHead
                    + headId * tokensPerBlock * dimsPerHead;
                T const* vInputPtr = inputBlockPtr + (layerId * 2 + 1) * headNum * tokensPerBlock * dimsPerHead
                    + headId * tokensPerBlock * dimsPerHead;

                int outputCacheIdx = headId / headNumDomainTP * DomainPPSize + layerId / layerNumDomainPP;
                T* outputCachePtr = outputCaches[outputCacheIdx];
                int layerIdInDomainPP = layerId % layerNumDomainPP;

                int headIdInDomainTP = headId % headNumDomainTP;
                T* kOutputPtr = outputCachePtr
                    + blockId * (layerNumDomainPP * 2 * headNumDomainTP * tokensPerBlock * dimsPerHead)
                    + layerIdInDomainPP * 2 * headNumDomainTP * tokensPerBlock * dimsPerHead
                    + headIdInDomainTP * tokensPerBlock * dimsPerHead;

                T* vOutputPtr = kOutputPtr + headNumDomainTP * tokensPerBlock * dimsPerHead;
#pragma unroll 1

                for (int tokenId = subWarpIdInGroup; tokenId < tokensPerBlock; tokenId += subWarpNumInGroup)
                {
                    T const* iKPtr = kInputPtr + tokenId * dimsPerHead;
                    T const* iVPtr = vInputPtr + tokenId * dimsPerHead;
                    T* oKPtr = kOutputPtr + tokenId * dimsPerHead;
                    T* oVPtr = vOutputPtr + tokenId * dimsPerHead;
#pragma unroll 1

                    for (int channelId = laneId * numElePerThread; channelId < dimsPerHead;
                         channelId += (subWarpSize * numElePerThread))
                    {
                        common::copy<vecSizeByte>(iKPtr + channelId, oKPtr + channelId);
                        common::copy<vecSizeByte>(iVPtr + channelId, oVPtr + channelId);
                    }
                }
            }
        }
    }
}

template <typename T, int subWarpSize, int subWarpNumInGroup, int vecSizeByte>
__global__ void concatenateKVCacheKernel(T const** __restrict__ inputCaches, T** __restrict__ outputBlocks,
    int tokensPerBlock, int numLayers, int headNum, int dimsPerHead, int outputBlockNum, int DomainPPSize,
    int DomainTPSize, int layerNumDomainPP, int headNumDomainTP)
{
    int const subWarpId = threadIdx.x / subWarpSize;
    int const laneId = threadIdx.x % subWarpSize;
    int const subWarpNum = blockDim.x / subWarpSize;
    int const subWarpGroupId = subWarpId / subWarpNumInGroup; //
    int const subWarpGroupNum = subWarpNum / subWarpNumInGroup;
    int const subWarpIdInGroup = subWarpId % subWarpNumInGroup;
    static_assert(vecSizeByte >= sizeof(T));
    int constexpr numElePerThread = vecSizeByte / sizeof(T);
    using VecType = typename common::BytesToType<vecSizeByte>::type;
#pragma unroll 1
    for (int blockId = blockIdx.y; blockId < outputBlockNum; blockId += gridDim.y)
    {
#pragma unroll 1
        for (int layerId = blockIdx.x; layerId < numLayers; layerId += gridDim.x)
        {

#pragma unroll 1
            for (int headId = subWarpGroupId; headId < headNum; headId += subWarpGroupNum)
            {

                T* outputBlockPtr = outputBlocks[blockId];
                T* kOutputPtr = outputBlockPtr + layerId * 2 * headNum * tokensPerBlock * dimsPerHead
                    + headId * tokensPerBlock * dimsPerHead;
                T* vOutputPtr = outputBlockPtr + (layerId * 2 + 1) * headNum * tokensPerBlock * dimsPerHead
                    + headId * tokensPerBlock * dimsPerHead;

                int inputCacheIdx = headId / headNumDomainTP * DomainPPSize + layerId / layerNumDomainPP;
                T const* inputCachePtr = inputCaches[inputCacheIdx];
                int layerIdInDomainPP = layerId % layerNumDomainPP;

                int headIdInDomainTP = headId % headNumDomainTP;
                T const* kInputPtr = inputCachePtr
                    + blockId * (layerNumDomainPP * 2 * headNumDomainTP * tokensPerBlock * dimsPerHead)
                    + layerIdInDomainPP * 2 * headNumDomainTP * tokensPerBlock * dimsPerHead
                    + headIdInDomainTP * tokensPerBlock * dimsPerHead;

                T const* vInputPtr = kInputPtr + headNumDomainTP * tokensPerBlock * dimsPerHead;
#pragma unroll 1
                for (int tokenId = subWarpIdInGroup; tokenId < tokensPerBlock; tokenId += subWarpNumInGroup)
                {
                    T const* iKPtr = kInputPtr + tokenId * dimsPerHead;
                    T const* iVPtr = vInputPtr + tokenId * dimsPerHead;
                    T* oKPtr = kOutputPtr + tokenId * dimsPerHead;
                    T* oVPtr = vOutputPtr + tokenId * dimsPerHead;

#pragma unroll 1
                    for (int channelId = laneId * numElePerThread; channelId < dimsPerHead;
                         channelId += (subWarpSize * numElePerThread))
                    {
                        common::copy<vecSizeByte>(iKPtr + channelId, oKPtr + channelId);
                        common::copy<vecSizeByte>(iVPtr + channelId, oVPtr + channelId);
                    }
                }
            }
        }
    }
}

template <typename T>
void splitKVCache(std::vector<runtime::ITensor::SharedPtr> const& kVCacheBlocks,
    std::vector<runtime::ITensor::SharedPtr>& outputSplitBlocks, kv_cache::CacheState const& iCacheState,
    kv_cache::CacheState const& oCacheState, int selfIdx, runtime::BufferManager const& bufferManager)
{

    auto inputBlockNum = kVCacheBlocks.size();
    auto targetRankInfo = targetIRanks(iCacheState, oCacheState, selfIdx);
    TLLM_CHECK(targetRankInfo.mIRanks.size()
        == (static_cast<size_t>(targetRankInfo.mDomainPPSize * targetRankInfo.mDomainTPSize)));
    auto outputCacheNum = targetRankInfo.mIRanks.size();
    TLLM_CHECK(outputCacheNum == outputSplitBlocks.size());
    TLLM_CHECK(inputBlockNum > 0);
    auto cacheBlockSize = kVCacheBlocks.at(0)->getSize();
    auto cacheDataType = kVCacheBlocks.at(0)->getDataType();
    std::vector<T*> CachePtrs;

    for (auto&& kvCacheBlock : kVCacheBlocks)
    {
        TLLM_CHECK(kvCacheBlock->getDataType() == cacheDataType);
        TLLM_CHECK(kvCacheBlock->getSize() == cacheBlockSize);
        CachePtrs.push_back(reinterpret_cast<T*>(kvCacheBlock->data()));
    }

    for (auto&& outputSplitBlock : outputSplitBlocks)
    {
        TLLM_CHECK(outputSplitBlock->getDataType() == cacheDataType);
        TLLM_CHECK(outputSplitBlock->getSize() == cacheBlockSize * inputBlockNum / outputCacheNum);
        CachePtrs.push_back(reinterpret_cast<T*>(outputSplitBlock->data()));
    }
    runtime::BufferManager::IBufferPtr PtrsDeviceBuffer
        = bufferManager.gpu(CachePtrs.size(), nvinfer1::DataType::kINT64);
    TLLM_CHECK(PtrsDeviceBuffer->getSizeInBytes() == CachePtrs.size() * sizeof(T*));
    bufferManager.copy(CachePtrs.data(), *PtrsDeviceBuffer, runtime::MemoryType::kCPU);

    constexpr int subWarpSize = 8;
    constexpr int subWarpNumInGroup = 8;
    int blockDimx = 128;

    int oPpNum = oCacheState.getParallelConfig().mPipelineParallelism;
    // layers
    unsigned int gridDimx = oCacheState.getModelConfig().mNbKvHeadsPerLayer.size() / oPpNum;
    // blockNum
    unsigned int gridDimy = inputBlockNum;

    dim3 gridDim{gridDimx, gridDimy};

    int const sizePerHead = oCacheState.getModelConfig().mSizePerHead;
    T const** inputBlockPtrsDev = reinterpret_cast<T const**>(PtrsDeviceBuffer->data());
    T** outputCachePtrsDev = reinterpret_cast<T**>(PtrsDeviceBuffer->data()) + inputBlockNum;
    int tokensPerBlock = oCacheState.getModelConfig().mTokensPerBlock;
    int numLayers = oCacheState.getModelConfig().mNbKvHeadsPerLayer.size() / oPpNum;
    int headNum = oCacheState.getModelConfig().mNbKvHeadsPerLayer[0];
    int dimsPerHead = oCacheState.getModelConfig().mSizePerHead;
    int DomainPPSize = targetRankInfo.mDomainPPSize;
    int DomainTPSize = targetRankInfo.mDomainTPSize;
    int iPPNum = iCacheState.getParallelConfig().mPipelineParallelism;
    int iTPNum = iCacheState.getParallelConfig().mTensorParallelism;
    int oTPNum = oCacheState.getParallelConfig().mTensorParallelism;
    int layerNumDomainPP = numLayers / DomainPPSize;
    int headNumDomainTP = headNum / DomainTPSize;

    TLLM_LOG_DEBUG(
        "splitKVCache: numLayers: %d, headNum: %d, DomainPPSize:%d, DomainTPSize:%d, layerNumDomainPP:%d, "
        "headNumDomainTP:%d",
        numLayers, headNum, DomainPPSize, DomainTPSize, layerNumDomainPP, headNumDomainTP);

    int const remainder = sizePerHead * sizeof(T) % 16;
    switch (remainder)
    {
    case 0:
    {
        splitKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 16>
            <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputBlockPtrsDev, outputCachePtrsDev,
                tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum, DomainPPSize, DomainTPSize,
                layerNumDomainPP, headNumDomainTP);
        break;
    }
    case 8:
    {
        splitKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 8>
            <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputBlockPtrsDev, outputCachePtrsDev,
                tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum, DomainPPSize, DomainTPSize,
                layerNumDomainPP, headNumDomainTP);
        break;
    }
    case 4:
    case 12:
    {
        if constexpr (sizeof(T) <= 4)
        {
            splitKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 4>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputBlockPtrsDev, outputCachePtrsDev,
                    tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum, DomainPPSize, DomainTPSize,
                    layerNumDomainPP, headNumDomainTP);
            break;
        }
    }

    case 2:
    case 6:
    case 10:
    case 14:
    {
        if constexpr (sizeof(T) <= 2)
        {
            splitKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 2>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputBlockPtrsDev, outputCachePtrsDev,
                    tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum, DomainPPSize, DomainTPSize,
                    layerNumDomainPP, headNumDomainTP);
            break;
        }
    }
    default:
    {
        if constexpr (sizeof(T) <= 1)
        {
            splitKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 1>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputBlockPtrsDev, outputCachePtrsDev,
                    tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum, DomainPPSize, DomainTPSize,
                    layerNumDomainPP, headNumDomainTP);
            break;
        }
        else
        {
            TLLM_THROW(" splitKVCacheDispatch no support data type error");
        }
    }
    }
}

void splitKVCacheDispatch(std::vector<runtime::ITensor::SharedPtr> const& kVCacheBlocks,
    std::vector<runtime::ITensor::SharedPtr>& ouputSplitBlocks, kv_cache::CacheState const& iCacheState,
    kv_cache::CacheState const& oCacheState, int selfIdx, runtime::BufferManager const& bufferManager)
{
    auto dataType = kVCacheBlocks.at(0)->getDataType();
    auto dataSize = tensorrt_llm::common::getDTypeSize(dataType);
    switch (dataSize)
    {
    case 8:
    {
        splitKVCache<double>(kVCacheBlocks, ouputSplitBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    case 4:
    {
        splitKVCache<float>(kVCacheBlocks, ouputSplitBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    case 2:
    {
        splitKVCache<half>(kVCacheBlocks, ouputSplitBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    case 1:
    {
        splitKVCache<uint8_t>(kVCacheBlocks, ouputSplitBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    default:
    {
        TLLM_THROW(" splitKVCacheDispatch no support data type error");
    }
    }
}

template <typename T>
void concatenateKVCache(std::vector<runtime::ITensor::SharedPtr> const& inputSplitBlocks,
    std::vector<runtime::ITensor::SharedPtr>& outputKvCacheBlocks,

    kv_cache::CacheState const& iCacheState, kv_cache::CacheState const& oCacheState, int selfIdx,
    runtime::BufferManager const& bufferManager)
{

    auto outputBlockNum = outputKvCacheBlocks.size();
    auto targetRankInfo = targetIRanks(iCacheState, oCacheState, selfIdx);
    TLLM_CHECK(targetRankInfo.mIRanks.size()
        == (static_cast<size_t>(targetRankInfo.mDomainPPSize * targetRankInfo.mDomainTPSize)));

    auto inputCacheNum = targetRankInfo.mIRanks.size();
    TLLM_CHECK(inputCacheNum == inputSplitBlocks.size());
    TLLM_CHECK(outputBlockNum > 0);
    auto cacheBlockSize = outputKvCacheBlocks.at(0)->getSize();
    auto cacheDataType = outputKvCacheBlocks.at(0)->getDataType();
    std::vector<T*> CachePtrs;
    for (auto&& kvCacheBlock : outputKvCacheBlocks)
    {
        TLLM_CHECK(kvCacheBlock->getDataType() == cacheDataType);
        TLLM_CHECK(kvCacheBlock->getSize() == cacheBlockSize);
        CachePtrs.push_back(reinterpret_cast<T*>(kvCacheBlock->data()));
    }
    for (auto&& inputSplitBlock : inputSplitBlocks)
    {
        TLLM_CHECK(inputSplitBlock->getDataType() == cacheDataType);
        TLLM_CHECK(inputSplitBlock->getSize() == cacheBlockSize * outputBlockNum / inputCacheNum);
        CachePtrs.push_back(reinterpret_cast<T*>(inputSplitBlock->data()));
    }
    runtime::BufferManager::IBufferPtr PtrsDeviceBuffer
        = bufferManager.gpu(CachePtrs.size(), nvinfer1::DataType::kINT64);
    TLLM_CHECK(PtrsDeviceBuffer->getSizeInBytes() == CachePtrs.size() * sizeof(T*));
    bufferManager.copy(CachePtrs.data(), *PtrsDeviceBuffer, runtime::MemoryType::kCPU);

    constexpr int subWarpSize = 8;
    constexpr int subWarpNumInGroup = 8;
    int blockDimx = 128;

    int oPpNum = oCacheState.getParallelConfig().mPipelineParallelism;
    // layers
    unsigned int gridDimx = oCacheState.getModelConfig().mNbKvHeadsPerLayer.size() / oPpNum;
    // blockNum
    unsigned int gridDimy = outputBlockNum;

    dim3 gridDim{gridDimx, gridDimy};
    int const sizePerHead = oCacheState.getModelConfig().mSizePerHead;
    int endLayerId = oCacheState.getModelConfig().mNbKvHeadsPerLayer.size() / oPpNum;
    T** ouptutBlockPtrsDev = reinterpret_cast<T**>(PtrsDeviceBuffer->data());
    T const** inputSplitBlockPtrsDev = reinterpret_cast<T const**>(PtrsDeviceBuffer->data()) + outputBlockNum;
    int tokensPerBlock = oCacheState.getModelConfig().mTokensPerBlock;
    int numLayers = oCacheState.getModelConfig().mNbKvHeadsPerLayer.size() / oPpNum;
    int headNum = oCacheState.getModelConfig().mNbKvHeadsPerLayer[0];
    int dimsPerHead = oCacheState.getModelConfig().mSizePerHead;
    int DomainPPSize = targetRankInfo.mDomainPPSize;
    int DomainTPSize = targetRankInfo.mDomainTPSize;
    int iPPNum = iCacheState.getParallelConfig().mPipelineParallelism;
    int iTPNum = iCacheState.getParallelConfig().mTensorParallelism;
    int oTPNum = oCacheState.getParallelConfig().mTensorParallelism;
    int layerNumDomainPP = numLayers / DomainPPSize;
    int headNumDomainTP = headNum / DomainTPSize;

    TLLM_LOG_DEBUG(
        "concatenateKVCache: numLayers: %d, headNum: %d, DomainPPSize:%d, DomainTPSize:%d, layerNumDomainPP:%d, "
        "headNumDomainTP:%d",
        numLayers, headNum, DomainPPSize, DomainTPSize, layerNumDomainPP, headNumDomainTP);
    int const remainder = sizePerHead * sizeof(T) % 16;

    switch (remainder)
    {
    case 0:
    {
        concatenateKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 16>
            <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev, ouptutBlockPtrsDev,
                tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum, DomainPPSize, DomainTPSize,
                layerNumDomainPP, headNumDomainTP);

        break;
    }
    case 8:
    {
        concatenateKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 8>
            <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev, ouptutBlockPtrsDev,
                tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum, DomainPPSize, DomainTPSize,
                layerNumDomainPP, headNumDomainTP);

        break;
    }
    case 4:
    case 12:
    {
        if constexpr (sizeof(T) <= 4)
        {
            concatenateKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 4>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev, ouptutBlockPtrsDev,
                    tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum, DomainPPSize, DomainTPSize,
                    layerNumDomainPP, headNumDomainTP);

            break;
        }
    }
    case 2:
    case 6:
    case 10:
    case 14:
    {
        if constexpr (sizeof(T) <= 2)
        {
            concatenateKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 2>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev, ouptutBlockPtrsDev,
                    tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum, DomainPPSize, DomainTPSize,
                    layerNumDomainPP, headNumDomainTP);

            break;
        }
    }
    default:
    {
        if constexpr (sizeof(T) <= 1)
        {
            concatenateKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 1>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev, ouptutBlockPtrsDev,
                    tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum, DomainPPSize, DomainTPSize,
                    layerNumDomainPP, headNumDomainTP);

            break;
        }
        else
        {
            TLLM_THROW(" concatenateKVCache no support data type error");
        }
    }
    }
}

void concatenateKvCacheV2Dispatch(std::vector<runtime::ITensor::SharedPtr> const& inputSplitBlocks,
    std::vector<runtime::ITensor::SharedPtr>& outputKvCacheBlocks, kv_cache::CacheState const& iCacheState,
    kv_cache::CacheState const& oCacheState, int selfIdx, runtime::BufferManager const& bufferManager)
{

    auto dataType = outputKvCacheBlocks.at(0)->getDataType();
    auto dataSize = tensorrt_llm::common::getDTypeSize(dataType);
    switch (dataSize)
    {
    case 8:
    {
        concatenateKVCache<double>(
            inputSplitBlocks, outputKvCacheBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    case 4:
    {
        concatenateKVCache<float>(
            inputSplitBlocks, outputKvCacheBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    case 2:
    {
        concatenateKVCache<half>(
            inputSplitBlocks, outputKvCacheBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    case 1:
    {
        concatenateKVCache<uint8_t>(
            inputSplitBlocks, outputKvCacheBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    default:
    {
        TLLM_THROW(" concatenateKVCache no support data type error");
    }
    }
}

} // namespace tensorrt_llm::executor::kv_cache
